#include "hip/hip_runtime.h"
// https://managedcuda.codeplex.com/documentation




#include <>
#include "float.h"
#include <builtin_types.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math.h>

#include "cutil_math.h"


#define F3ADDASIGNI(A,B,I) F3ADDASIGN(A[I],B[I])
#define F3ADDASIGN(A,B) A.x+=B.x; A.y+=B.y; A.z+=B.z; 

#define F3ADD(A,B) make_float3(A.x+B.x, A.y+B.y, A.z+B.z)

#define F3MULTF(A,B) make_float3(A.x*B, A.y*B, A.z*B)


extern "C"
{


	__constant__ __device__ float randomFloats[1024];


	__global__ void updateParticles(
		int* randomIndex, float deltaTime, int length,
		float3* currentPositionH, float3* currentVelocityH, float3* currentAccelerationH, float* currentLifeTimeH, float4* startColorH, float4* endColorH, float* startSizeH, float* endSizeH, float* startLifeTimeH
	)
	{
		unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
		unsigned int tidMax = blockDim.x * gridDim.x;

		while (tid < length) 
		{	
			if (currentLifeTimeH[tid]>=0) {
				currentLifeTimeH[tid] -= deltaTime;
				F3ADDASIGNI(currentVelocityH, currentAccelerationH, tid)
				F3ADDASIGNI(currentPositionH, currentVelocityH, tid)
			}
			tid += tidMax;
		}
	}



	inline __host__ __device__ float getRandomFloat(int* randomIndex) {
		int old = atomicAdd(randomIndex, 1);
		if (old > 1000) {
			*randomIndex = 0;
			old = 0;
		}
		return randomFloats[old];
	}

	__global__ void generateParticles(
		int* randomIndex, int length,
		float3* currentPositionH, float3* currentVelocityH, float3* currentAccelerationH, float* currentLifeTimeH, float4* startColorH, float4* endColorH, float* startSizeH, float* endSizeH, float* startLifeTimeH,
		int* realCount, int* desiredCount, float3 aroundPosition, float4 startColor, float4 endColor, float startSize, float endSize, float startLifeTime
	)
	{
		unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
		unsigned int tidMax = blockDim.x * gridDim.x;

		while (tid < length)
		{
			if (currentLifeTimeH[tid] <= 0.0f) 
			{
				int old = *desiredCount;
				if (old>0) {
					old = atomicSub(desiredCount, 1);
					if (old > 0) {
						//atomicAdd(realCount, 1);

#define RF getRandomFloat(randomIndex)

						//float3 direction = make_float3(0,0,0);
						float3 direction = normalize(make_float3(RF - 0.5, RF - 0.5, RF - 0.5));
						float r = RF;
						direction = F3MULTF(direction, r);

						//currentPositionH[tid] = F3ADD(aroundPosition,direction);
						currentPositionH[tid] = aroundPosition;
						currentVelocityH[tid] = direction;
						//currentVelocityH[tid] = make_float3(0, 0, 0);
						//r = -0.5*RF;
						//currentAccelerationH[tid] = F3MULTF(direction, r);
						currentAccelerationH[tid] = make_float3(0, 0, 0);
						currentLifeTimeH[tid] = startLifeTime;
						startColorH[tid] = startColor;
						endColorH[tid] = endColor;
						startSizeH[tid] = startSize;
						endSizeH[tid] = endSize;
						startLifeTimeH[tid] = startLifeTime;
					}
				}
			}
			tid += tidMax;
		}
	}
}


int main()
{
	return 0;
}

